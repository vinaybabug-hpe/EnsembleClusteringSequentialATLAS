#include "labels.h"

namespace kmeans {
namespace detail {

struct cublas_state {
    hipblasHandle_t cublas_handle;
    cublas_state() {
        hipblasStatus_t stat;
        stat = hipblasCreate(&cublas_handle);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS initialization failed" << std::endl;
            exit(1);
        }
    }
    ~cublas_state() {
        hipblasStatus_t stat;
        stat = hipblasDestroy(cublas_handle);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS destruction failed" << std::endl;
            exit(1);
        }
    }
};


cublas_state state;

void gemm(hipblasOperation_t transa, hipblasOperation_t transb,
          int m, int n, int k, const float *alpha,
          const float *A, int lda, const float *B, int ldb,
          const float *beta,
          float *C, int ldc) {
    hipblasStatus_t status = hipblasSgemm(state.cublas_handle, transa, transb,
                                        m, n, k, alpha,
                                        A, lda, B, ldb,
                                        beta,
                                        C, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Invalid Sgemm" << std::endl;
        exit(1);
    }
}

void gemm(hipblasOperation_t transa, hipblasOperation_t transb,
          int m, int n, int k, const double *alpha,
          const double *A, int lda, const double *B, int ldb,
          const double *beta,
          double *C, int ldc) {
    hipblasStatus_t status = hipblasDgemm(state.cublas_handle, transa, transb,
                                        m, n, k, alpha,
                                        A, lda, B, ldb,
                                        beta, 
                                        C, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Invalid Dgemm" << std::endl;
        exit(1);
    }
}

}
}
